#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <hipblas.h>
#include "../include/util.h"
#include "kernel.h"

#define N_RUNS 10


int  main (int argc, char** argv) {

	ErrChk(hipSetDevice(0));

	if(argc<2){
		printf("Usage: input the batch size\n");
		exit(EXIT_FAILURE);
	}

	int BATCH = atoi(argv[1]);
	
	int *M;
	int *N;
	int *K;

	M = (int*) malloc(BATCH * sizeof(int));
	N = (int*) malloc(BATCH * sizeof(int));
	K = (int*) malloc(BATCH * sizeof(int));

	std::fstream fs;
	fs.open("../data/input");
	if (!fs.is_open()){
		printf("Error opening input\n");
		exit(EXIT_FAILURE);
	}
	
	//read matrix config	
	for (int i=0; i<BATCH; ++i){
		fs>>M[i]>>N[i]>>K[i];
	}

    float **A;
	float **B;
	float **C;

	A = (float**) malloc(BATCH * sizeof(float*));
	B = (float**) malloc(BATCH * sizeof(float*));
	C = (float**) malloc(BATCH * sizeof(float*));

	for (int i=0; i<BATCH; ++i){
		ErrChk(hipMalloc((void**)&A[i], M[i]*K[i]*sizeof(float)));
		ErrChk(hipMalloc((void**)&B[i], K[i]*N[i]*sizeof(float)));
		ErrChk(hipMalloc((void**)&C[i], M[i]*N[i]*sizeof(float)));
	}

	float **dev_A;
	float **dev_B;
	float **dev_C;

    ErrChk(hipMalloc((void**)&dev_A, BATCH*sizeof(float*)));
    ErrChk(hipMalloc((void**)&dev_B, BATCH*sizeof(float*)));
    ErrChk(hipMalloc((void**)&dev_C, BATCH*sizeof(float*)));

	ErrChk(hipMemcpy(dev_A, A, BATCH*sizeof(float*), hipMemcpyHostToDevice));
	ErrChk(hipMemcpy(dev_B, B, BATCH*sizeof(float*), hipMemcpyHostToDevice));
	ErrChk(hipMemcpy(dev_C, C, BATCH*sizeof(float*), hipMemcpyHostToDevice));


	int *dev_M, *dev_N, *dev_K;
	ErrChk(hipMalloc((void**)&dev_M, BATCH*sizeof(int)));
	ErrChk(hipMalloc((void**)&dev_N, BATCH*sizeof(int)));
	ErrChk(hipMalloc((void**)&dev_K, BATCH*sizeof(int)));

	ErrChk(hipMemcpy(dev_M, M, BATCH*sizeof(int), hipMemcpyHostToDevice));
	ErrChk(hipMemcpy(dev_N, N, BATCH*sizeof(int), hipMemcpyHostToDevice));
	ErrChk(hipMemcpy(dev_K, K, BATCH*sizeof(int), hipMemcpyHostToDevice));

	float elapsedTime = 0.f;
    double time=0.f;
	float gflops_per_sec = 0.f;
	double gflops = 0.f;
	for (int i=0; i<BATCH; ++i)
		gflops += ((2 * int64_t(M[i]) * int64_t(N[i]) * int64_t(K[i])) + (2 * int64_t(M[i]) * int64_t(N[i])) ) / 1.0e9;
	hipEvent_t start, stop;

    dim3 block_size;
    block_size.x = 64;
    block_size.y = 1;
	block_size.z = 1;

    dim3 grid_size;
    grid_size.x = M[0] / 16;
    grid_size.y = N[0] / 16;
	grid_size.z = BATCH;

	for (int j=1; j<BATCH; ++j){
		grid_size.x = (grid_size.x > M[j]/16)?(grid_size.x):(M[j]/16);
		grid_size.y = (grid_size.y > N[j]/16)?(grid_size.y):(N[j]/16);
	}

	//warm-up
	gemm<64, 16, 16><<<grid_size, block_size, sizeof(float)*2*16*16>>>(dev_M, dev_N, dev_K, dev_A, dev_B, dev_C);
	KernelErrChk();

	ErrChk(hipEventCreate(&start));
	ErrChk(hipEventRecord(start,0));

	for (int run = 0; run<N_RUNS; ++run){
		gemm<64, 16, 16><<<grid_size, block_size, sizeof(float)*2*16*16>>>(dev_M, dev_N, dev_K, dev_A, dev_B, dev_C);
		KernelErrChk();
	}

	ErrChk(hipEventCreate(&stop));
	ErrChk(hipEventRecord(stop,0));
	ErrChk(hipEventSynchronize(stop));
	ErrChk(hipEventElapsedTime(&elapsedTime, start,stop));

	time = elapsedTime/N_RUNS;
	time /= 1.0e3; //convert time unit from millisecond to second
	gflops_per_sec   = gflops / time;
	printf("%f\n", gflops_per_sec);

	for (int i=0; i<BATCH; ++i){
		ErrChk(hipFree(A[i]));		
		ErrChk(hipFree(B[i]));		
		ErrChk(hipFree(C[i]));		
	}

	free(M);
	free(N);
	free(K);
	free(A);
	free(B);
	free(C);
	ErrChk(hipFree(dev_M));		
	ErrChk(hipFree(dev_N));		
	ErrChk(hipFree(dev_K));		
	ErrChk(hipFree(dev_A));		
	ErrChk(hipFree(dev_B));		
	ErrChk(hipFree(dev_C));		

	return 0;
	
}
